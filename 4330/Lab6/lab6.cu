/* Author: Ross Bearden
   Instructor: Dr. Pettey
   Class: CSCI 4330
   Date: 05/03/17

   Purpose: This program will calculate the sum of two matrices using GPUs and the cuda 
            language to do the calculations
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ 
void vectorAdd(
  int* vector1,
  int* vector2,
  int* vectorResult)
{
  vectorResult[threadIdx.x] =
    vector1[threadIdx.x] +
    vector2[threadIdx.x];
}

int main(int argc, char* argv[])
{
  int rows = 16;
  int columns = 32;
  int i,j;
  int firstArray[rows][columns];
  int secondArray[rows][columns];

  //Read the matrices
  for(i = 0; i < rows; i++){
    for(j = 0; j < columns; j++){
      scanf("%d", &firstArray[i][j]);
    }
  }

  for(i = 0; i < rows; i++){
    for(j = 0; j < columns; j++){
      scanf("%d", &secondArray[i][j]);
    }
  }
/*
  for(i = 0; i < rows; i++){
    for(j = 0; j < columns; j++){
      printf("%d", firstArray[i][j]);
    }
  }

  for(i = 0; i < rows; i++){
    for(j = 0; j < columns; j++){
      printf("%d", secondArray[i][j]);
    }
  }
  */

  int* vector1;
  int* vector2;
  int sizeOfVector = 512;
  int* deviceResults;
  int hostResults[512];

  hipMalloc((void**) &vector1, sizeof(int) * sizeOfVector);
  hipMalloc((void**) &vector2, sizeof(int) * sizeOfVector);
  hipMalloc((void**) &deviceResults, sizeof(int) * sizeOfVector);
  

  hipMemcpy(vector1, firstArray, sizeof(int) * sizeOfVector,
    hipMemcpyHostToDevice);
  hipMemcpy(vector2, secondArray, sizeof(int) * sizeOfVector,
    hipMemcpyHostToDevice);
  vectorAdd<<<1, sizeOfVector>>> (vector1, vector2, deviceResults);
  hipMemcpy(hostResults, deviceResults, sizeof(int) * sizeOfVector,
    hipMemcpyDeviceToHost);

  for(i = 0; i < rows; i++){
   for(j = 0; j < columns; j++){
    printf("%i  ", hostResults[j]);
    }
   printf("\n");
  }

  hipFree(vector1);
  hipFree(vector2);
  hipFree(deviceResults);

  return 0;
}
